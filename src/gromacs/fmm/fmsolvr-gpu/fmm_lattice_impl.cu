#include "hip/hip_runtime.h"
#include "fmm.hpp"
#include "cuda_LATTICE.hpp"
#include "cuda_lib.hpp"

namespace gmx_gpu_fmm{

template <typename CoefficientMatrix>
__global__
void __reset_omega(CoefficientMatrix **omega, size_t p1xp2_2)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < p1xp2_2)
        omega[0]->flush(i);
}

void fmm_algorithm::lattice_impl(){

    const int num_of_streams = STREAMS;
    typedef typename CoeffMatrix::value_type complex_type;

    //wait for m2l
    for(size_t i = 0; i < STREAMS; ++i)
    {
        hipStreamWaitEvent(priority_streams[current_priority_stream], priority_events[i], 0 );
    }

    size_t op_p1xx2 = ( (2*p+1) * (2*p+1) );

    if(dipole_compensation)
    {
        if(depth == 0)
        {
            __reset_omega<CoeffMatrix><<<(p1xp2_2-1)/512 + 1, 512, 0, priority_streams[current_priority_stream]>>>(omega, p1xp2_2);
        }

        int last_stream = current_priority_stream;
        __P2M_P2L_dipole_corr<<<1, 64, 0, priority_streams[current_priority_stream]>>>(&q0abc[0], omega, &fake_particles[0], mu, p, fake_particle_size);
        hipEventRecord(priority_events[current_priority_stream],priority_streams[current_priority_stream]);
        current_priority_stream = (++current_priority_stream)%num_of_streams;

        hipStreamWaitEvent(priority_streams[current_priority_stream], priority_events[last_stream], 0 );
        __AoS_addto_SoA_omega__(box, omegaSoA, 1, p1xp2_2, priority_streams[current_priority_stream]);
        hipStreamWaitEvent(priority_streams[current_priority_stream], priority_events[last_stream], 0 );
        __AoS_addto_SoA_mu__(box, muSoA, 1, p1xp2_2, priority_streams[current_priority_stream]);
    }

    if (!open_boundary_conditions)
    {

#ifndef GMX_FMM_DOUBLE
        dim3 grid(1,1,1);
        dim3 block(p1+1,p1,1);
        __lattice<CoeffMatrix, CoeffMatrixSoA, Box, Real, Real3, complex_type>
        <<<grid,block,(p1*p1+op_p1xx2)*sizeof(complex_type), priority_streams[current_priority_stream]>>>
        (box, omegaSoA, muSoA, Lattice, 0, num_boxes_tree, p, p1, p1xx2, op_p1xx2, lattice_rescale);
#else
        if(p<21)
        {
            dim3 grid(1,1,1);
            dim3 block(p1+1,p1,1);
            __lattice<CoeffMatrix, CoeffMatrixSoA, Box, Real, Real3, complex_type>
            <<<grid,block,(p1*p1+op_p1xx2)*sizeof(complex_type), priority_streams[current_priority_stream]>>>
            (box, omegaSoA, muSoA, Lattice, 0, num_boxes_tree, p, p1, p1xx2, op_p1xx2, lattice_rescale);
        }
        else
        {
            dim3 griD(p1,1,1);
            dim3 blocK(p1,1,1);
            __lattice_no_shared<CoeffMatrix, CoeffMatrixSoA, Box, Real, Real3, complex_type>
            <<<griD,blocK,0,priority_streams[current_priority_stream]>>>
            (box, omegaSoA, muSoA, Lattice, 0, num_boxes_tree, p, p1, p1xx2, op_p1xx2, lattice_rescale);
        }
#endif
    }
}

}//namespace end
